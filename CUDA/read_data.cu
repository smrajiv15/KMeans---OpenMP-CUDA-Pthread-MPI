/*
 * Data reading function for Parallel Computing class, 
 * Assignment: K-Means Algorithm (CUDA)
 *
 * To students: You should not modify this file
 * 
 * Author:
 *     Wei Wang <wei.wang@utsa.edu>
 */
#ifndef _GNU_SOURCE
#define _GNU_SOURCE
#endif
#include <stdio.h>
#include <stdlib.h>
#include <err.h>

#include "k_means.h"

/*
 * read data points from input file
 */
int read_points_from_file(char *data_file, struct point *pts, int *m)
{
	FILE *fp;
	size_t len = 0;
	ssize_t read = 0;
	char *line = NULL;
	int ret = 0;

	/* open the file */
	fp = fopen(data_file, "r");
	if(fp == NULL)
		err(-1, "Unable to open file");

	*m = 0;

	/* read in the coordinates of the points */
	while((read = getline(&line, &len, fp)) != -1){
		ret = sscanf(line, "%lf,%lf\n", &(pts[*m].x),
			     &(pts[*m].y));
		if(ret != 2)
			continue;

		/* increase the size of pts if necessary */
		(*m)++;
		if(*m == MAX_POINTS){
			printf("Too many data points (maximum %d points)\n",
			       MAX_POINTS);
			exit(-1);
		}
	 }
	
	fclose(fp);

	if(line)
		free(line);



#ifdef DEBUG
	{
		int i;
		for(i = 0; i < *m; i++)
			printf("%lf,%lf\n", pts[i].x, pts[i].y);
	}
		
#endif
      

	return 0;
}

#include "hip/hip_runtime.h"

/*
 * Skeleton function for Parallel Computing Class, 
 * Assignment: K-Means Algorithm (CUDA)
 *
 * To students: You should finish the implementation of k_means algorithm.
 *              You should add device functions/kernels to perform k_means on 
 *              GPU. The "k_means" function in this file is just an interface
 *              for passing in basic parameters needed.. You need to add GPU 
 *              kernels and launch them in the "k_means" function.
 *
 *              Note that the "k_means" function has two input parameters for
 *              block count and thread count per block. Please use these two
 *              parameters when launching your kernels.
 * 
 * Author:
 *     Wei Wang <wei.wang@utsa.edu>
 */
#include <stdio.h>
#include <float.h>
#include <math.h>

#include "k_means.h"

#define CHECK(call)\
{\
	const hipError_t error = call; \
	if(error != hipSuccess) { \
		printf("Error: %s: Line: %d\n", __FILE__, __LINE__);\
		printf("Code : %d - Reason : %s\n", error, hipGetErrorString(error));\
		exit(1);\
	}\
}

#define DIST(p1, p2) ((p2.x - p1.x) * (p2.x - p1.x) + (p2.y - p1.y) * (p2.y - p1.y))

struct thread_limits {
	int start_point;
	int end_point;
	int tot_pts;
	double sumx;
	double sumy;
	unsigned clust_size_pt;
};


__host__ void assign_thread_limits(struct thread_limits *tl, int tt, int num_pts) {
	float pts_per_thread = ceil(num_pts / (float)tt); 
	int max_pts = num_pts;
	int i;
	
	for(i = 0; i < tt; i++) {
		if((max_pts - pts_per_thread) >= pts_per_thread && max_pts > 0) {
			tl[i].tot_pts = pts_per_thread;
			tl[i].end_point = max_pts;	
			max_pts -= pts_per_thread;
			tl[i].start_point = max_pts;
		} else {
			tl[i].start_point = 0;
			tl[i].end_point = max_pts;
			tl[i].tot_pts = max_pts;
			max_pts = 0;
			break;
		}
	}
}


__global__ void cluster_identifier(struct point *p, struct point *u, int *c, \
																        struct thread_limits *tl, int m, int k) {
	int i, j;
	double dist;
	double min_dist;
	int t_id = blockIdx.x * blockDim.x + threadIdx.x;	

	if(tl[t_id].tot_pts > 0) {
		for(i = tl[t_id].start_point; i < tl[t_id].end_point; i++) {
			min_dist = DBL_MAX;
			for(j = 0; j < k; j++) {				
				dist = DIST(p[i], u[j]);

				if(dist < min_dist) {
					min_dist = dist;
					c[i] = j;
				}		
			}
		}
	} 
}


__global__ void compute_center(int ci, int m, struct point *p, \
																				int *c, struct thread_limits *tl) {
	int i;	
	int t_id = blockIdx.x * blockDim.x + threadIdx.x;	

	//computing per thread sum w.r.t the cluster	
	if(tl[t_id].tot_pts > 0) {
		for(i = tl[t_id].start_point; i < tl[t_id].end_point; i++) {		
			if(c[i] == ci) {
				tl[t_id].sumx += p[i].x;
				tl[t_id].sumy += p[i].y;
				tl[t_id].clust_size_pt += 1;
			}
		}
	}
}


/*
 * k_means: k_means clustering algorithm implementation.
 *
 * Input parameters:
 *     struct point p[]: array of data points
 *     int m           : number of data points in p[]
 *     int k           : number of clusters to find
 *     int iters       : number of clustering iterations to run
 *     int block_cnt   : number of blocks to use
 *     int threads_per_block: number of threads per block
 *
 * Output parameters:   
 *     struct point u[]: array of cluster centers
 *     int c[]         : cluster id for each data points
 */
void k_means(struct point p[MAX_POINTS], 
	     int m, 
	     int k,
	     int iters,
	     struct point u[MAX_CENTERS],
	     int c[MAX_POINTS],
	     int block_cnt,
	     int threads_per_block)
{
	int i, j, l;
	//Device Memory
	struct point *d_p;
	struct point *d_u;
	int *d_c;
	int mc = m;
	struct thread_limits *d_tl;
	double sumx, sumy;
	long clust_size;
	int tot_threads = block_cnt * threads_per_block;
	struct thread_limits tl[tot_threads];

	memset(tl, 0, sizeof(struct thread_limits) * tot_threads);
	assign_thread_limits(tl, tot_threads, m); 
	
	/* randomly initialized the centers */
	for(j = 0; j < k; j++)
		u[j] = random_center(p);
	
	hipMalloc(&d_p, sizeof(struct point) * MAX_POINTS);
	hipMalloc(&d_u, sizeof(struct point) * MAX_CENTERS);
	hipMalloc(&d_c, sizeof(int) * MAX_POINTS);
	hipMalloc(&d_tl,sizeof(struct thread_limits) * tot_threads); 
	
	CHECK(hipMemcpy(d_p, p, sizeof(struct point) * MAX_POINTS, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_u, u, sizeof(struct point) * MAX_CENTERS, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_c, c, sizeof(int) * MAX_POINTS, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_tl, tl, sizeof(struct thread_limits) * tot_threads, hipMemcpyHostToDevice));

	for(i = 0; i < iters; i++) {
		cluster_identifier<<<block_cnt, threads_per_block>>>(d_p, d_u, d_c, d_tl, m, k);	
	
		for(j = 0; j < k; j++) {
			sumx = 0;
			sumy = 0;
			clust_size = 0;

			compute_center<<<block_cnt, threads_per_block>>>(j, m, d_p, d_c, d_tl); 
			
			CHECK(hipMemcpy(tl, d_tl, sizeof(struct thread_limits) * tot_threads, hipMemcpyDeviceToHost));

			for(l = 0; l < tot_threads; l++) {
				if(tl[l].tot_pts > 0) {
					sumx += tl[l].sumx;
					sumy += tl[l].sumy;
					clust_size += tl[l].clust_size_pt;
					tl[l].sumx = 0;
					tl[l].sumy = 0;
					tl[l].clust_size_pt = 0;
				}
			}

			if(clust_size > 0) {
				u[j].x = sumx / (double)clust_size;
				u[j].y = sumy / (double) clust_size;
			} else {
				u[j] = random_center(p);
			}
			CHECK(hipMemcpy(d_tl, tl, sizeof(struct thread_limits) * tot_threads, hipMemcpyHostToDevice));
		}	
		CHECK(hipMemcpy(d_u, u, sizeof(struct point) * MAX_CENTERS, hipMemcpyHostToDevice));
	}
	
	hipFree(d_p);
	hipFree(d_u);
	hipFree(d_c);
	hipFree(d_tl);	

  return;
}
